/****************************************************************************
 *
 * cuda-reduction2.cu - Reduction with CUDA atomic operations
 *
 * This version works for any array length n; however, it still
 * requires that BLKDIM is a power of two.
 *
 * Copyright (C) 2019 by Moreno Marzolla <moreno.marzolla(at)unibo.it>
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *    http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 * ---------------------------------------------------------------------------
 *
 * This program realizes a not-so-simple sum-reduction on the
 * GPU. Each thread block copies a portion of the array in shared
 * memory; then, all threads within the same block cooperate to
 * compute the sum of the local data by organizing the computation as
 * a tree. Atomic operations are used to complete the execution
 * using the GPU only.
 *
 * Compile with:
 * nvcc cuda-reduction3.cu -o cuda-reduction3
 *
 * Run with:
 * ./cuda-reduction3
 *
 ****************************************************************************/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

/* For this program to work, BLKDIM must be a power of two */
#define BLKDIM 1024

/* Note: *result must be initially zero for this kernel to work! */
__global__ void sum( int *a, int n, int *result )
{
    __shared__ int temp[BLKDIM];
    int lindex = threadIdx.x;
    int gindex = threadIdx.x + blockIdx.x * blockDim.x;
    int bsize = blockDim.x / 2;

    if ( gindex < n ) {
        temp[lindex] = a[gindex];
    } else {
        temp[lindex] = 0;
    }

    /* wait for all threads to finish the copy operation */
    __syncthreads(); 

    /* All threads within the block cooperate to compute the local sum */
    while ( bsize > 0 ) {
        if ( lindex < bsize ) {
            temp[lindex] += temp[lindex + bsize];
        }
        bsize = bsize / 2; 
        /* threads must synchronize before performing the next
           reduction step */
        __syncthreads(); 
    }

    if ( 0 == lindex ) {
        atomicAdd(result, temp[0]);
    }
}

void init( int *v, int n )
{
    int i;
    for (i=0; i<n; i++) {
        v[i] = 2;
    }
}

int main( int argc, char *argv[] ) 
{
    int *h_a, result = 0;
    int *d_a, *d_result;
    int n = 1024*512;
    
    assert( (BLKDIM & (BLKDIM-1)) == 0 ); /* check if BLKDIM is a power of two using the "bit hack" from http://www.graphics.stanford.edu/~seander/bithacks.html#DetermineIfPowerOf2 */

    if ( argc > 1 ) {
        n = atoi(argv[1]);
    }

    const size_t size = n * sizeof(*h_a);
    const int n_of_blocks = (n + BLKDIM - 1)/BLKDIM;

    /* Allocate space for host copies of array */
    h_a = (int*)malloc(size);
    init(h_a, n);

    /* Allocate space for device copies of aarray */
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_result, sizeof(*d_result));

    /* Copy inputs to device */
    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    /* Copy the initial result (zero) to the device; this is important
       since the kernel requires *d_result to be initially zero. */
    hipMemcpy(d_result, &result, sizeof(result), hipMemcpyHostToDevice);

    /* Launch sum() kernel on the GPU */
    sum<<<n_of_blocks, BLKDIM>>>(d_a, n, d_result);

    /* Copy the result from device memory to host memory */
    hipMemcpy(&result, d_result, sizeof(result), hipMemcpyDeviceToHost);

    /* Check result */
    const int expected = 2*n;
    if ( result != expected ) {
        printf("Check FAILED: got %d, expected %d\n", result, expected);
    } else {
        printf("Check OK: sum = %d\n", result);
    }
    /* Cleanup */
    free(h_a);
    hipFree(d_a); hipFree(d_result);
    return EXIT_SUCCESS;
}
